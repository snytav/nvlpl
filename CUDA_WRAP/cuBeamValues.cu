#include "hip/hip_runtime.h"
#include "../run_control.h"
#include <stdio.h>
#include "labels.h"
#include "cuParticles.h"

#include <string>
using namespace std;

int CUDA_WRAP_alloc_beam_values(int Np,int num_attr,double **h_p,double **d_p)
{
        int err;
#ifdef CUDA_WRAP_CHECK_BEAM_VALUES_ALLOWED 

        puts("in alloc values");
	
	*h_p   = (double*) malloc(num_attr*Np*sizeof(double));
	
	err = hipMalloc((void**)d_p,num_attr*Np*sizeof(double));
	printf("alloc values error %d \n",err);
	
	hipMemset(*d_p,0,num_attr*Np*sizeof(double));
	
	memset(*h_p,0,num_attr*Np*sizeof(double));

	int err1 = hipGetLastError();

	printf("end alloc values error %d\n",err1);
#endif
	
	return err;
}

double printAttributesTable(double *h_copy,double *h_p,int Np,int num_attr,char *beam_or_plasma,int nstep)
{
	FILE *f,*f_out,*f_dump;
	double cu_x,x;
	char fname_attr[1000];
	char name_out[100];
	char name_dump[100];
	int wrong_particles;
	double frac_rude = 0.0,frac_medium = 0.0,frac_light = 0.0,frac;

	sprintf(fname_attr,"attr_%s_nstep_%010d.dat",beam_or_plasma,nstep);
	if((f = fopen(fname_attr,"wt")) == NULL) return -1.0;
	
//	double wrong_array = (double *)malloc(num_attr*sizeof(double));
//	double delta_array = (double *)malloc(num_attr*sizeof(double));
//	
//	wrong_array = (double *)malloc(num_attr*sizeof(double));
//	delta_array = (double *)malloc(num_attr*sizeof(double));
//        int width = Ny*Nz; 
//        double *h_data_in;
	

	for(int n = 0;n < num_attr; n++)
        {
	   for (int i = 0;i < Np;i++)
	   {
	
              cu_x = h_copy[i*num_attr + n];
     	      x    = h_p   [i*num_attr + n];
     	   
     	      if(fabs(cu_x - x) > PARTICLE_TOLERANCE)
     	      {
	         wrong_particles++;
	      }
		   
	   }
           frac = (((double)wrong_particles)/Np)*100;
           fprintf(f,"attribute %3d wrong particles %10d of %10d, %2f  \n",n,wrong_particles,Np,frac);

	if(frac < 1.0)
        {
	  frac_light += 1.0;
	}
		else
		{
			if (frac < 30)
			{
				frac_medium += 1.0;
			}
			else
			{
			    frac_rude += 1.0;
			}
		}
	}
	frac_light  /= num_attr;
	frac_medium /= num_attr;
	frac_rude   /= num_attr;
	fprintf(f,"attributes error light %10.3e medium %10.3e rude %10.3e \n ",frac_light,frac_medium,frac_rude);
	fclose(f);
	
	return frac_rude;
}

double CUDA_WRAP_check_beam_values(int Np,int num_attr,double *h_p,double *d_p,int blocksize_x,int blocksize_y,char *fname,char *beam_or_plasma,int nstep)
{
        int cell_number,wrong_particles = 0;
	double    *h_copy,frac_err,delta = 0.0,*wrong_array,*delta_array,res;
	int wrong_flag = 0;
	printf("in ceck beam values %s \n ",beam_or_plasma);
	
	FILE *f,*f_out,*f_dump;
	char fname_attr[1000];
	char name_out[100];
	char name_dump[100];

	

	sprintf(name_out,"%s_nstep_%010d.dat",beam_or_plasma,nstep);
	sprintf(name_dump,"VLPL_CPU_values_%s_nstep_%010d.dat",beam_or_plasma,nstep);
	printf("fname_attr %s out %s dump %s \n",fname_attr, fname,name_out,name_dump);
	
	//if((f = fopen(fname_attr,"wt")) == NULL) return -1.0;
	f_out = fopen(name_out,"wt");
	if((f_dump = fopen(name_dump,"wt")) == NULL) return -1.0;
	printf("files out %s dump %s  opened  \n", name_out,name_dump);
	
	wrong_array = (double *)malloc(num_attr*sizeof(double));
	delta_array = (double *)malloc(num_attr*sizeof(double));
//        int width = Ny*Nz; 
//        double *h_data_in;
	
	printf("BEGIN  %s-RELATED VALUES sCHECK =============================================================================",beam_or_plasma);
	
	//part_per_cell_max = findMaxNumberOfParticlesPerCell(mesh,i_layer,Ny,Nz,p_CellArray);
	h_copy   = (double*) malloc(num_attr*Np*sizeof(double));
	
	//GET PARTICLE DATA FROM SURFACE
	//CUDA_WRAP_get_particle_surface(partSurfOut,cuOutputArrayX,NUMBER_ATTRIBUTES*part_per_cell_max,width,h_data_in);
	int err = hipMemcpy(h_copy,d_p,num_attr*Np*sizeof(double),hipMemcpyDeviceToHost);

	for(int i  = 0;i < 10;i++)
        {
	    int n = 0;	
            printf(" GPU particles  i %d n %d POS %d h_plasma_values %e \n",i,n,
                        POSITION(i,n,num_attr) ,
                        h_copy [  POSITION(i,n,num_attr)      ]
                        );
        }

	printf("d_p[0] %e d_p[9] %e\n ",h_copy[0],h_copy[9]);
	if((res = printAttributesTable(h_copy,h_p,Np,num_attr,beam_or_plasma,nstep)) < 0.0) return -1.0;
        int Np1 = Np;
	string s = "";
	fprintf(f_out,"%15s ",s.c_str());
	for (int i = 0;i < Np1;i++)
	{
	    fprintf(f_out,"%15d ",i);
	}
	fprintf(f_out,"\n");
        for(int n = 0;n < num_attr; n++)
        {
           int wpa = 0,wrong_particles = 0;;
	   double fr_attr,x,cu_x;
	
	   for (int i = 0;i < Np1;i++)
           {
	
            cu_x = h_copy[ POSITION(i,n,num_attr)    ];
     	    x    = h_p   [ POSITION(i,n,num_attr)    ];
			  
            fprintf(f_dump,"attr %10d np  %10d cpu %25.15e gpu %25.15e delta %15.5e \n",n,i,x,cu_x,fabs(cu_x - x));
           }
       }
       fclose(f_dump);	
       return res;
   }
   
/*			  
      	       
#ifdef CUDA_WRAP_PARTICLE_VALUES_DETAILS	     
#endif	     
        }
        fprintf(f_out,"\n");
		fprintf(f_dump,"\n");
        fr_attr = (double)wpa/(Np);
        fprintf(f,"value %3d OK %7.2f wrong %7.2f delta %15.5e wpa %10d Np %10d CORRECT %10d \n",n,1.0 - fr_attr,fr_attr,delta,wpa,Np,Np - wpa);
	//printf("\n value %3d OK %7.2f wrong %7.2f delta %15.5e wpa %10d Np %10d CORRECT %10d \n",n,1.0 - fr_attr,fr_attr,delta,wpa,Np,Np - wpa);
	if(Np - wpa < blocksize_x*blocksize_y) wrong_flag = 1;
	
	wrong_array[n] = fr_attr;
	delta_array[n] = delta;
	
	//puts("___________________________________________________________________________________________________________");
  *///  }

/*	free(h_copy);
	
	frac_err = (double)wrong_particles/(Np*num_attr);
	fclose(f_out);
	
	
/*	FILE *wf,*df;
	if(write_values_first_call == 1)
	{
	   if((wf = fopen("values_wrong.dat","wt")) == NULL) return 1;
	   if((df = fopen("values_delta.dat","wt")) == NULL) return 1;
	   
	   write_values_first_call = 0;
	}
        else
	{
	   if((wf = fopen("values_wrong.dat","at")) == NULL) return 1;
	   if((df = fopen("values_delta.dat","at")) == NULL) return 1;
	}
	
	fprintf(wf,"Layer %5d ",iLayer);
	fprintf(df,"Layer %5d ",iLayer);
	if(iLayer <= 477)
	{
	   int ig45 = 0;
	}*/
       /*
	double max_delta = 0.0;
	for(int i = 0;i < num_attr;i++)
	{
//	    fprintf(wf,"%15.5e ",wrong_array[i]);
//	    fprintf(df,"%15.5e ",delta_array[i]);
	    
	    if(max_delta < delta_array[i]) 
	    {
	       max_delta = delta_array[i];
//	       last_max_delta_value = i;
	    }
	    
	      
	}
//	fprintf(wf,"\n");
//	fprintf(df,"\n");
	
//	fclose(wf);
//	fclose(df);
  
          
//	free(wrong_array);
//	free(delta_array);
	
/*	last_wrong = frac_err;
	last_delta = max_delta;
	
        if(wrong_flag == 1) printf("\nONE OR MORE VALUES ARE WRONG !!!!!!!!!!!!!!!!!!!!!!!!!\n");
	printf("%s-RELATED CHECK OK %.4f wrong %.4f delta %15.5e =================================================\n",beam_or_plasma,
	       1.0-frac_err,frac_err,max_delta);
	fclose(f);
	
        return frac_err;
*/	
//}
