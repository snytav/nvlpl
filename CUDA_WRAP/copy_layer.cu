#include<stdlib.h>
#include "../cell3d.h"
#include "../mesh.h"
#include "cuLayers.h"
#include "cuBeam.h"

int CUDA_WRAP_get_particles_number(Mesh *mesh,Cell *p_CellArray)
{
   double *Ex,*Ey,*Ez,*Bx,*By,*Bz,*Jx,*Jy,*Jz,*Rho;
//   beamParticle *bp;
   cudaLayer *h_dl;
   int np = 0,Ny,Nz;

   int err = hipGetLastError();
   printf("in getLayerParticles begin err %d \n",err);

   Ny = mesh->GetMy();
   Nz = mesh->GetMz();
   for (int k=0; k<Nz; k++)
   {
      for (int j=0; j<Ny; j++)
      {
          long ncc = mesh->GetNyz(j,  k);
          Cell &ccc = p_CellArray[ncc];

	      Particle *p  = ccc.GetParticles();

	      for(;p;np++)
	      {
		  p = p->p_Next;
	      }

      }
   }

   return np;

}

int Particle2beamParticle(beamParticle *bp,Particle *p,int i,int j,int k)
{
    bp->f_X      = p->f_X;
    bp->f_Y      = p->f_Y;
    bp->f_Z      = p->f_Z;
    bp->f_Px     = p->f_Px;
    bp->f_Py     = p->f_Py, 
    bp->f_Pz     = p->f_Pz, 
    bp->f_Weight = p->f_Weight; 
    bp->f_Q2m    = p->f_Q2m;
    bp->isort    = p->i_Sort;
    bp->i_X      = i;
    bp->i_Y      = j;
    bp->i_Z      = k;

	return 0;
}

int copyParticleHost2Host(beamParticle *p1,beamParticle *p2)
{
   p1->f_Px     = p2->f_Px;
   p1->f_Py     = p2->f_Py;
   p1->f_Pz     = p2->f_Pz;
   p1->f_Q2m    = p2->f_Q2m;
   p1->f_Weight = p2->f_Weight;
   p1->f_X      = p2->f_X;
   p1->f_Y      = p2->f_Y;
   p1->f_Z      = p2->f_Z;
   p1->f_Pz     = p2->f_Pz;
   p1->i_X      = p2->i_X;
   p1->i_Y      = p2->i_Y;
   p1->i_Z      = p2->i_Z;

  p1->isort    = p2->isort;


}

int LayerAlloc(cudaLayer **cl,int Ny,int Nz, int Np)
{
    *cl = (cudaLayer*)malloc(sizeof(cudaLayer));

    (*cl)->Ny = Ny;
    (*cl)->Nz = Nz;
    (*cl)->Ny = Ny;
    (*cl)->Np = Np;
    (*cl)->particles = (beamParticle *)malloc(Np*sizeof(beamParticle));

    int size = Ny*Nz*sizeof(double);
    (*cl)->Ex = (double *)malloc(size);
    (*cl)->Ey = (double *)malloc(size);
    (*cl)->Ez = (double *)malloc(size);

    (*cl)->Bx = (double *)malloc(size);
    (*cl)->By = (double *)malloc(size);
    (*cl)->Bz = (double *)malloc(size);

    (*cl)->Jx = (double *)malloc(size);
    (*cl)->Jy = (double *)malloc(size);
    (*cl)->Jz = (double *)malloc(size);

    (*cl)->Rho = (double *)malloc(size);
    (*cl)->JxBeam = (double *)malloc(size);
    (*cl)->RhoBeam = (double *)malloc(size);
    (*cl)->fftJxBeamHydro = (double *)malloc(size);

    (*cl)->fftRhoBeamHydro = (double *)malloc(size);





    return 0;
}

int CUDA_WRAP_copy_from_CellArray2Layer(Mesh *mesh,Cell *p_CellArray,cudaLayer **cl,int iLayer)
{
   double *Ex,*Ey,*Ez,*Bx,*By,*Bz,*Jx,*Jy,*Jz,*Rho;
   beamParticle *bp;
   cudaLayer *h_dl;
   int Ny,Nz,Np;
   CUDA_MALLOC_TEST("Cell2Layer begin"); 






   Ny = mesh->GetMy();
   Nz = mesh->GetMz();

   Np = CUDA_WRAP_get_particles_number(mesh,p_CellArray);
   Np = 40960;
   printf(" Ny %d Nz %d Np %d\n",Ny,Nz,Np);
    
   LayerAlloc(cl,Ny,Nz,Np);
   CUDA_MALLOC_TEST("LayerAlloc");
   int np = 0;

   long Mx,My,Mz,dMx,dMy,dMz;
   mesh->GetSizes(Mx,My,Mz,dMx,dMy,dMz);
   for (int k=0; k<Mz; k++)
   {
      for (int j=0; j<My; j++)
      {
          long ncc = mesh->GetNyz(j,  k);
          Cell &ccc = p_CellArray[ncc];

	      Particle *p  = ccc.GetParticles();

          for(;p;np++)
	      {
		     p = p->p_Next;
		     if(p != NULL)
		     {
				beamParticle bp;
				Particle2beamParticle(&bp,p,iLayer,j,k);
                copyParticleHost2Host(&((*cl)->particles[np]),&bp);
             }
	      }

	  int n =  get2D_index(*cl,k,j); 
         // add copy arrays!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
	  //printf("k %5d j %5d ncc %10d Ny*Nz %10d \n",k,j,n,Ny*Nz);
         (*cl)->Ex[n] = ccc.GetEx();
         (*cl)->Ey[n] = ccc.GetEy();
         (*cl)->Ez[n] = ccc.GetEz();




      }
   }
  // exit(0);
   CUDA_MALLOC_TEST("END 2aRRAY");

   return 0;

}
